#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "ArgMax.hpp"
#include <hip/hip_fp16.h>
#include <cassert>

// TODO: Move this to a common header
inline bool is_CHW(nvinfer1::Dims const& dims) {
  return (dims.nbDims == 3 &&
          dims.type[0] == nvinfer1::DimensionType::kCHANNEL &&
          dims.type[1] == nvinfer1::DimensionType::kSPATIAL &&
          dims.type[2] == nvinfer1::DimensionType::kSPATIAL);
}

nvinfer1::Dims ArgMaxPlugin::getOutputDimensions(int index,
                                                        const nvinfer1::Dims *inputDims,
                                                        int nbInputs) {
  assert(nbInputs == 1);
  nvinfer1::Dims const& input = inputDims[0];
  assert(is_CHW(input));
  assert(_ndims == 2);
  assert(index == 0);
  nvinfer1::Dims output;
  output.nbDims = input.nbDims;
  int s = 0;
  for( int d=0; d<input.nbDims; ++d ) {
    output.type[d] = input.type[d];
    if( input.type[d] != nvinfer1::DimensionType::kSPATIAL ) {
      output.d[d] = 1;
    } else {
      output.d[d] = input.d[d];
    }
  }
  return output;
}

int ArgMaxPlugin::initialize() {
  _output_dims = this->getOutputDimensions(0, &this->getInputDims(0), 1);
  assert(is_CHW(this->getInputDims(0)));
  assert(is_CHW(_output_dims));
  assert(_ndims == 2);
  return 0;
}

template <typename Data>
__global__
void argmax_kernel(int nbatch,
                  int c,
                  int2 osize,
                  Data const* idata, int istride, int ibatchstride,
                  int*       odata, int ostride, int obatchstride)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if ((x >= osize.x*osize.y) || (x < 0)) return;
    float max = -1e6;
    int am = 0;
    for (int i = 0; i < c; i++)
        if (tensor[x*c+i]>=max) {
        max = tensor[x*c+i];
        am = i;
    }
    result[x]=am;
}

template <typename Data>
__global__
void resize_bilinear_kernel_2d(int nbatch,
                              float2 scale,
                              int2 osize,
                              Data const* idata, int istride, int ibatchstride,
                              Data*       odata, int ostride, int obatchstride) {
  int x0 = threadIdx.x + blockIdx.x * blockDim.x;
  int y0 = threadIdx.y + blockIdx.y * blockDim.y;
  int z0 = blockIdx.z;
  int src_cols = int(osize.x / scale.x);
  int src_rows = int(osize.y / scale.y);
  for( int batch=z0; batch<nbatch; batch+=gridDim.z ) {
    for( int oy=y0; oy<osize.y; oy+=blockDim.y*gridDim.y ) {
      for( int ox=x0; ox<osize.x; ox+=blockDim.x*gridDim.x ) {
        float src_x = ox / scale.x;
        float src_y = oy / scale.y;
        int x1 = int(src_x);
        int y1 = int(src_y);
        int x2 = src_x + 1;
        int y2 = src_y + 1;
        int x2_read = ::min(x2, src_cols - 1);
        int y2_read = ::min(y2, src_rows - 1);

        float src_reg = idata[batch * ibatchstride + y1 * istride + x1];
        odata[batch * obatchstride + oy * ostride + ox] = src_reg * ((x2 - src_x) * (y2 - src_y));

        src_reg = (float)(idata[batch * ibatchstride + y1 * istride + x2_read]);
        odata[batch * obatchstride + oy * ostride + ox] = (float)odata[batch * obatchstride + oy * ostride + ox] + src_reg * (float)((src_x - x1) * (y2 - src_y));

        src_reg = (float)(idata[batch * ibatchstride + y2_read * istride + x1]);
        odata[batch * obatchstride + oy * ostride + ox] = (float)odata[batch * obatchstride + oy * ostride + ox] + src_reg * (float)((x2 - src_x) * (src_y - y1));

        src_reg = (float)(idata[batch * ibatchstride + y2_read * istride + x2_read]);
        odata[batch * obatchstride + oy * ostride + ox] = (float)(odata[batch * obatchstride + oy * ostride + ox]) + src_reg * (float)((src_x - x1) * (src_y - y1));

      }
    }
  }
}

int ArgMaxPlugin::enqueue(int batchSize,
                                 const void *const *inputs, void **outputs,
                                 void *workspace, hipStream_t stream) {
  auto const& input_dims = this->getInputDims(0);
  int nchan = input_dims.d[0];
  switch( _ndims ) {
  case 2: {
    float2 scale = {_scale[1], _scale[0]};
    int2 osize = {_output_dims.d[2], _output_dims.d[1]};
    int istride =   input_dims.d[2];
    int ostride = _output_dims.d[2];
    int ibatchstride =   input_dims.d[1] * istride;
    int obatchstride = _output_dims.d[1] * ostride;
    dim3 block(32, 16);
    dim3 grid((osize.x - 1) / block.x + 1,
              (osize.y - 1) / block.y + 1,
              std::min(batchSize * nchan, 65535));

      //  std::cout << "bilinear" << std::endl;
        if (getDataType()==nvinfer1::DataType::kFLOAT) {
            resize_bilinear_kernel_2d<<<grid, block, 0, stream>>>
                                                               (batchSize * nchan, scale, osize,
                                                                static_cast<float const*>( inputs[0]), istride, ibatchstride,
                    static_cast<float*      >(outputs[0]), ostride, obatchstride);
        } else {
            resize_bilinear_kernel_2d<<<grid, block, 0, stream>>>
                                                               (batchSize * nchan, scale, osize,
                                                                static_cast<__half const*>( inputs[0]), istride, ibatchstride,
                    static_cast<__half*      >(outputs[0]), ostride, obatchstride);
        }

    return hipGetLastError() != hipSuccess;
  }
  default: return -1;
  }
}
